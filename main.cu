#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <random>
#include "nn.cuh"
#include "timer.h"

int main()
{
    auto nn = MNISTNeuralNetwork();
    start_timer();
    nn.learn(1);
    end_timer();
    return 0;
    
}
