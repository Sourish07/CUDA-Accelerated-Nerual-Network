#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <random>
#include "nn.cuh"
#include "timer.h"

int main()
{
    auto nn = MNISTNeuralNetwork();
    
    auto timer = Timer("Network-GPU");
    nn.learn(5);
    timer.stop();

    return 0;
}
